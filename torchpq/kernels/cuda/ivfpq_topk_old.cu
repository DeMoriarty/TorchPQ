#include "hip/hip_runtime.h"
#define _VOLATILE_ 
#define likely(x)      __builtin_expect(!!(x), 1)
#define unlikely(x)    __builtin_expect(!!(x), 0)
#define load(x)        __ldcg(x)
#define store(x, value) __stcs(x, value)
#ifndef INFINITY
#define INFINITY __int_as_float(0x7f800000)
#endif

typedef unsigned char uint8_t;
typedef long long ll_t;

typedef struct __device_builtin__ __builtin_align__(_NCS_)
{
  uint8_t _VARNAMES_;
} _uint8n_t;

typedef union {
  _uint8n_t u8n;
  uint8_t val[_NCS_];
} uint8n_t;


__device__ __forceinline__ float atomicMax(float *address, float val)
{
  int ret = __float_as_int(*address);
  while(val > __int_as_float(ret))
  {
    int old = ret;
    if((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
        break;
  }
  return __int_as_float(ret);
}

__device__ void load_precomputed(
  const float *precomputed,
  _VOLATILE_ float *sMem,
  int nQuery
){
  const int tid = threadIdx.x;
  const int qid = blockIdx.x;
  if (tid < 256){
    #pragma unroll
    for (int i = 0; i < _M_; i++){
      #if _TPB_ >= 256
      int adr = (i * nQuery * _K_) + (qid * _K_) + (tid);
      sMem[i * _K_ + tid] = precomputed[adr];
      
      #else
      #pragma unroll
      for (int j = 0; j < _K_ / _TPB_; j++){
        int adr = (i * nQuery * _K_) + (qid * _K_) + (j * _TPB_ + tid);
        sMem[i * _K_ + j * _TPB_ + tid] = precomputed[adr];
      }
      #endif
    }
  }
  __syncthreads();
}

__device__ __forceinline__ unsigned int bfe(
  unsigned int source,
  unsigned int bitIndex
) {
  unsigned int bit;
  asm volatile("bfe.u32 %0, %1, %2, %3;" : "=r"(bit) : "r"((unsigned int) source), "r"(bitIndex), "r"(1));
  return bit;
}

__device__ __forceinline__ void warp_comparator(
  float &value,
  float &index,
  const int stride,
  const int direction
){
  const float otherValue = __shfl_xor_sync(0xFFFFFFFF, value, stride);
  const float otherIndex = __shfl_xor_sync(0xFFFFFFFF, index, stride);
  bool condition = value < otherValue == direction;
  index = condition ? otherIndex : index;
  value = condition ? otherValue : value;
}

__device__ __forceinline__ void block_comparator(
  float &value,
  float &index,
  const int stride,
  const int direction,
  const int laneID,
  _VOLATILE_ float sMem[]
){
  float tempPrecomputed1 = sMem[laneID];
  float tempPrecomputed2 = sMem[_TPB_ + laneID];
  __syncthreads();

  sMem[laneID] = value;
  sMem[_TPB_ + laneID] = index;
  __syncthreads();

  float otherValue = sMem[laneID ^ stride];
  float otherIndex = sMem[_TPB_ + laneID ^ stride];
  __syncthreads();

  sMem[laneID] = tempPrecomputed1;
  sMem[_TPB_ + laneID] = tempPrecomputed2;
  __syncthreads();

  bool condition = value < otherValue == direction;
  value = condition ? otherValue : value;
  index = condition ? otherIndex : index;
  /*
  */
}

__device__ __forceinline__ void block_comparator_noop(
){
  __syncthreads();
  __syncthreads();
  __syncthreads();
  __syncthreads();
}

__device__ __forceinline__ void thread_comparator(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  const int direction
){
  bool condition = value > otherValue == direction;
  if (condition){
    value = otherValue;
    index = otherIndex;
    /*
    value = value + otherValue;
    otherValue = value - otherValue;
    value = value - otherValue;

    index = index + otherIndex;
    otherIndex = index - otherIndex;
    index = index - otherIndex;
    */
  }
}

__device__ void bitonic_sort_2(
  float &value,
  float &index,
  int laneID
){
  warp_comparator(value, index, 1, bfe(laneID, 1) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_4(
  float &value,
  float &index,
  int laneID
){
  bitonic_sort_2(value, index, laneID);
  warp_comparator(value, index, 2, bfe(laneID, 2) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 2) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_8(
  float &value,
  float &index,
  int laneID
){
  bitonic_sort_4(value, index, laneID);
  warp_comparator(value, index, 4, bfe(laneID, 3) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 3) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 3) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_16(
  float &value,
  float &index,
  int laneID
){
  bitonic_sort_8(value, index, laneID);
  warp_comparator(value, index, 8, bfe(laneID, 4) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 4) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 4) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 4) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_32(
  float &value,
  float &index,
  int laneID
){
  bitonic_sort_16(value, index, laneID);
  warp_comparator(value, index, 16, bfe(laneID, 5) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 5) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 5) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 5) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 5) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_global_2(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

__device__ void bitonic_sort_global_4(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

__device__ void bitonic_sort_global_8(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

__device__ void bitonic_sort_global_16(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

__device__ void bitonic_sort_global_32(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

#if _TPB_ >= 64
__device__ void bitonic_sort_64(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_32(value, index, laneID);
  block_comparator(value, index, 32, bfe(laneID, 6) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 6) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 6) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 6) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 6) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 6) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_64(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 64 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));

    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
  }
}


#if _TPB_ >= 128
__device__ void bitonic_sort_128(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_64(value, index, sMem, laneID);
  block_comparator(value, index, 64, bfe(laneID, 7) ^ bfe(laneID, 6), laneID, sMem);
  block_comparator(value, index, 32, bfe(laneID, 7) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 7) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 7) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 7) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 7) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 7) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_128(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 128 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 64, !bfe(laneID, 6), laneID, sMem);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
    block_comparator_noop();
  }
}


#if _TPB_ >= 256
__device__ void bitonic_sort_256(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_128(value, index, sMem, laneID);
  block_comparator(value, index, 128, bfe(laneID, 8) ^ bfe(laneID, 7), laneID, sMem);
  block_comparator(value, index, 64, bfe(laneID, 8) ^ bfe(laneID, 6), laneID, sMem);
  block_comparator(value, index, 32, bfe(laneID, 8) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 8) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 8) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 8) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 8) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 8) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_256(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 256 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 128, !bfe(laneID, 7), laneID, sMem);
    block_comparator(value, index, 64, !bfe(laneID, 6), laneID, sMem);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
  }
}

#if _TPB_ >= 512
__device__ void bitonic_sort_512(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_256(value, index, sMem, laneID);
  block_comparator(value, index, 256, bfe(laneID, 9) ^ bfe(laneID, 8), laneID, sMem);
  block_comparator(value, index, 128, bfe(laneID, 9) ^ bfe(laneID, 7), laneID, sMem);
  block_comparator(value, index, 64, bfe(laneID, 9) ^ bfe(laneID, 6), laneID, sMem);
  block_comparator(value, index, 32, bfe(laneID, 9) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 9) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 9) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 9) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 9) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 9) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_512(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 512 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 256, !bfe(laneID, 8), laneID, sMem);
    block_comparator(value, index, 128, !bfe(laneID, 7), laneID, sMem);
    block_comparator(value, index, 64, !bfe(laneID, 6), laneID, sMem);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
  }
}


#if _TPB_ >= 1024
__device__ void bitonic_sort_1024(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_512(value, index, sMem, laneID);
  block_comparator(value, index, 512, bfe(laneID, 10) ^ bfe(laneID, 9), laneID, sMem);
  block_comparator(value, index, 256, bfe(laneID, 10) ^ bfe(laneID, 8), laneID, sMem);
  block_comparator(value, index, 128, bfe(laneID, 10) ^ bfe(laneID, 7), laneID, sMem);
  block_comparator(value, index, 64, bfe(laneID, 10) ^ bfe(laneID, 6), laneID, sMem);
  block_comparator(value, index, 32, bfe(laneID, 10) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 10) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 10) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 10) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 10) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 10) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_1024(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 256 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 512, !bfe(laneID, 9), laneID, sMem);
    block_comparator(value, index, 256, !bfe(laneID, 8), laneID, sMem);
    block_comparator(value, index, 128, !bfe(laneID, 7), laneID, sMem);
    block_comparator(value, index, 64, !bfe(laneID, 6), laneID, sMem);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
  }
}


__device__ void load_consume_data(
  const uint8n_t* data,
  _VOLATILE_ float sMem[],
  float &value,
  int iN, int nData
){
  #pragma unroll
  for (int i = 0; i < _M_ / _NCS_; i++){
    uint8n_t threadData = data[(i * nData) + iN];
    float pre0 = sMem[(i * _NCS_ + 0) * _K_ + int(threadData.val[0]) ];
    float pre1 = sMem[(i * _NCS_ + 1) * _K_ + int(threadData.val[1]) ];
    float pre2 = sMem[(i * _NCS_ + 2) * _K_ + int(threadData.val[2]) ];
    float pre3 = sMem[(i * _NCS_ + 3) * _K_ + int(threadData.val[3]) ];
    value += pre0;
    value += pre1;
    value += pre2;
    value += pre3;
  }
}

__device__ void load_data(
  const uint8n_t* data,
  uint8n_t dataCache[_M_ / _NCS_],
  int iN, int nData
){
  #pragma unroll
  for (int i = 0; i < _M_ / _NCS_; i++){
    uint8n_t threadData = data[(i * nData) + iN];
    dataCache[i] = threadData;
  }
}

__device__ void consume_data(
  _VOLATILE_ float sMem[],
  uint8n_t dataCache[_M_ / _NCS_],
  float &value
){
  #pragma unroll
  for (int i = 0; i < _M_ / _NCS_; i++){
    uint8n_t threadData = dataCache[i];
    float pre0 = sMem[(i * _NCS_ + 0) * _K_ + int(threadData.val[0]) ];
    float pre1 = sMem[(i * _NCS_ + 1) * _K_ + int(threadData.val[1]) ];
    float pre2 = sMem[(i * _NCS_ + 2) * _K_ + int(threadData.val[2]) ];
    float pre3 = sMem[(i * _NCS_ + 3) * _K_ + int(threadData.val[3]) ];
    value += pre0;
    value += pre1;
    value += pre2;
    value += pre3;
  }
}

extern "C"
__global__ void ivfpq_topk(
  const uint8n_t* __restrict__ data,
  const float* __restrict__ precomputed,
  const uint8_t* __restrict__ isEmpty,
  const ll_t* __restrict__ cellStart,
  const ll_t* __restrict__ cellSize,
  const ll_t* __restrict__ totalSize,
  float* __restrict__ gValue,
  ll_t* __restrict__ gIndex,
  int nData, int nQuery, int nProbe, int nCandidates
) {
  const int tid = threadIdx.x; // thread ID
  const int qid = blockIdx.x; // query ID

  extern __shared__ _VOLATILE_ float sMem[]; // M * K
  load_precomputed(precomputed, sMem, nQuery);
  float finalValue = -654321;
  float finalIndex = -1;
  const ll_t threadTotalSize = totalSize[qid];
  const int nIter = (threadTotalSize + _TPB_ - 1) / _TPB_;
  int cCell = 0;
  int cCellStart = cellStart[qid * nProbe + cCell];
  int cCellSize = cellSize[qid * nProbe + cCell];
  int cCellEnd = cCellStart + cCellSize;
  int iN = cCellStart + tid;

  for (int i = 0; i < nIter; i++){
    while (iN >= cCellEnd){
      cCell ++;  // increment cell index by 1
      if (cCell >= nProbe)
        break;
      int pCellEnd = cCellEnd;
      cCellStart = cellStart[qid * nProbe + cCell];
      cCellSize = cellSize[qid * nProbe + cCell];
      cCellEnd = cCellStart + cCellSize;
      iN = iN - pCellEnd + cCellStart;
    }
    float value;
    float index = iN;
    int cIsEmpty = 0;
    if (cCellStart <= iN && iN < cCellEnd){
      value = 0.f;
      cIsEmpty = isEmpty[iN];
      //load_consume_data(data, sMem, value, iN, nData);

      uint8n_t dataCache[_M_ / _NCS_];
      load_data(data, dataCache, iN, nData);
      consume_data(sMem, dataCache, value);
      /*
      */
    } else {
      value = -123456.f;
    }
    value = cIsEmpty == 0 ? value : -987654.f;
    index = cIsEmpty == 0 ? index : -1;

    #if _TPB_ == 32
    bitonic_sort_32(value, index, tid);

    #elif _TPB_ == 64
    bitonic_sort_64(value, index, sMem, tid);

    #elif _TPB_ == 128
    bitonic_sort_128(value, index, sMem, tid);

    #elif _TPB_ == 256
    bitonic_sort_256(value, index, sMem, tid);

    #elif _TPB_ == 512
    bitonic_sort_512(value, index, sMem, tid);

    #elif _TPB_ == 1024
    bitonic_sort_1024(value, index, sMem, tid);
    #endif
    
    switch (nCandidates){
      case 2:
        bitonic_sort_global_2(
          finalValue, finalIndex, value, index,
          tid);
          break;
      case 4:
        bitonic_sort_global_4(
          finalValue, finalIndex, value, index,
          tid);
          break;
      case 8:
        bitonic_sort_global_8(
          finalValue, finalIndex, value, index,
          tid);
          break;
      case 16:
        bitonic_sort_global_16(
          finalValue, finalIndex, value, index,
          tid);
          break;
      case 32:
        bitonic_sort_global_32(
          finalValue, finalIndex, value, index,
          tid);
          break;
      case 64:
        bitonic_sort_global_64(
          finalValue, finalIndex, value, index,
          sMem, tid);
          break;
      case 128:
        bitonic_sort_global_128(
          finalValue, finalIndex, value, index,
          sMem, tid);
          break;
      case 256:
        bitonic_sort_global_256(
          finalValue, finalIndex, value, index,
          sMem, tid);
          break;
      case 512:
        bitonic_sort_global_512(
          finalValue, finalIndex, value, index,
          sMem, tid);
          break;
      case 1024:
        bitonic_sort_global_1024(
          finalValue, finalIndex, value, index,
          sMem, tid);
          break;
    }
    iN += _TPB_;
  }

  if (_TPB_ - nCandidates <= tid){
    const int writeAddress = (qid * nCandidates) + tid - ( _TPB_ - nCandidates);
    gValue[writeAddress] = finalValue;
    gIndex[writeAddress] = finalIndex;
  }
}
#include "hip/hip_runtime.h"
#define _VOLATILE_ 
#define likely(x)      __builtin_expect(!!(x), 1)
#define unlikely(x)    __builtin_expect(!!(x), 0)
#define load(x)        __ldcg(x)
#define store(x, value) __stcs(x, value)
#define isnan(x) ( x != x )
#define N_WARPS _TPB_/32
#ifndef INFINITY
#define INFINITY __int_as_float(0x7f800000)
#endif

typedef long long ll_t;
typedef struct __builtin_align__(8)
{
  float value;
  int index;
} pair;

#if (__CUDA_ARCH__ < 700)
__device__ void __nanosleep(unsigned int ns){
  clock_t start_clock = clock();
  clock_t clock_offset = 0;
  while (clock_offset < ns)
  {
    clock_offset = clock() - start_clock;
  }
}
#endif 


__device__ __forceinline__ unsigned int bfe(
  unsigned int source,
  unsigned int bitIndex
) {
  unsigned int bit;
  asm volatile("bfe.u32 %0, %1, %2, %3;" : "=r"(bit) : "r"((unsigned int) source), "r"(bitIndex), "r"(1));
  return bit;
}

__device__ __forceinline__ void warp_comparator(
  float &value,
  int &index,
  const int stride,
  const int direction
){
  const float otherValue = __shfl_xor_sync(0xFFFFFFFF, value, stride);
  const int otherIndex = __shfl_xor_sync(0xFFFFFFFF, index, stride);
  bool condition = value < otherValue == direction;
  index = condition ? otherIndex : index;
  value = condition ? otherValue : value;
}

__device__ __forceinline__ void block_comparator(
  float &value,
  int &index,
  const int stride,
  const int direction,
  const int laneID,
  _VOLATILE_ float valSmem[_TPB_],
  _VOLATILE_ int idxSmem[_TPB_]
){
  valSmem[laneID] = value;
  idxSmem[laneID] = index;
  __syncthreads();

  float otherValue = valSmem[laneID ^ stride];
  float otherIndex = idxSmem[laneID ^ stride];
  __syncthreads();

  bool condition = value < otherValue == direction;
  value = condition ? otherValue : value;
  index = condition ? otherIndex : index;
}

__device__ __forceinline__ void thread_comparator(
  float &value,
  int &index,
  float otherValue,
  int otherIndex,
  const int direction
){
  bool condition = value > otherValue == direction;
  if (condition){
    value = otherValue;
    index = otherIndex;
  }
}

__device__ __forceinline__ void bitonic_sort_2(
  float &value,
  int &index,
  int laneID
){
  warp_comparator(value, index, 1, bfe(laneID, 1) ^ bfe(laneID, 0));
}

__device__ __forceinline__ void bitonic_sort_4(
  float &value,
  int &index,
  int laneID
){
  bitonic_sort_2(value, index, laneID);
  warp_comparator(value, index, 2, bfe(laneID, 2) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 2) ^ bfe(laneID, 0));
}

__device__ __forceinline__ void bitonic_sort_8(
  float &value,
  int &index,
  int laneID
){
  bitonic_sort_4(value, index, laneID);
  warp_comparator(value, index, 4, bfe(laneID, 3) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 3) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 3) ^ bfe(laneID, 0));
}

__device__ __forceinline__ void bitonic_sort_16(
  float &value,
  int &index,
  int laneID
){
  bitonic_sort_8(value, index, laneID);
  warp_comparator(value, index, 8, bfe(laneID, 4) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 4) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 4) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 4) ^ bfe(laneID, 0));
}

__device__ __forceinline__ void bitonic_sort_32(
  float &value,
  int &index,
  int laneID
){
  bitonic_sort_16(value, index, laneID);
  warp_comparator(value, index, 16, bfe(laneID, 5) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 5) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 5) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 5) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 5) ^ bfe(laneID, 0));
}
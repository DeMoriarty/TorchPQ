#include "hip/hip_runtime.h"
#ifndef INFINITY
#define INFINITY __int_as_float(0x7f800000)
#endif


typedef long long ll_t;

extern "C"
__global__ void get_div_by_address(
  const ll_t* __restrict__ address,
  const ll_t* __restrict__ divStart,
  const ll_t* __restrict__ divEnd,
  ll_t* __restrict__ divs,
  int nAddress, int nDivs
) {
  int tid = threadIdx.x; // thread ID
  int bid = blockIdx.x; // block ID
  int aStartBlock = bid * _TPB_ * _TA_;
  // int aStartThread = aStartBlock + tid * _TA_;

  extern __shared__ ll_t smem[];  //[_TPB_ * _TA_]

  ll_t threadAddress[_TA_];
#pragma unroll
  for (int i=0; i<_TA_; i++){
    int aid = aStartBlock + i * _TPB_ + tid;
    if (aid < nAddress){
      threadAddress[i] = address[aid];
    } else {
      threadAddress[i] = -3;
    }
  }

#pragma unroll
  for (int i=0; i<_TA_; i++){
    int idx = i * _TPB_ + tid;
    smem[idx] = threadAddress[i];

  }
  __syncthreads();
  ll_t threadMax = -1;
#pragma unroll
  for (int i=0; i<_TA_; i++){
    int idx = tid * _TA_ + i;
    threadAddress[i] = smem[idx];
    threadMax = max(threadMax, threadAddress[i]);
  }
  ll_t threadMin = threadAddress[0];

  int nIters = __float2int_rn(__log2f(float(nDivs))) + 1;
  
  int leftPivotRange[2] = {0, nDivs};
  int leftPivot = 0;
  for (int i=0; i<nIters; i++){
    leftPivot = (leftPivotRange[0] + leftPivotRange[1]) / 2;
    ll_t leftDivStart = divStart[leftPivot];
    ll_t leftDivEnd = divEnd[leftPivot];
    if (leftDivStart <= threadMin && leftDivEnd > threadMin){
      break;
    } else if (leftDivStart > threadMin) {
      leftPivotRange[1] = leftPivot;
    } else if (leftDivEnd <= threadMin) {
      leftPivotRange[0] = leftPivot;
    }
  } // end for i

  int rightPivotRange[2] = {0, nDivs};
  int rightPivot = nDivs - 1;
  for (int i=0; i<nIters; i++){
    rightPivot = (rightPivotRange[0] + rightPivotRange[1]) / 2;
    ll_t rightDivStart = divStart[rightPivot];
    ll_t rightDivEnd = divEnd[rightPivot];
    if (rightDivStart <= threadMax && rightDivEnd > threadMax){
      break;
    } else if (rightDivStart > threadMax) {
      rightPivotRange[1] = rightPivot;
    } else if (rightDivEnd <= threadMax) {
      rightPivotRange[0] = rightPivot;
    }
  } // end for i

  for (int i=leftPivot; i<rightPivot + 1; i++){ // is +1 necessary?
    ll_t cDivStart = divStart[i];
    ll_t cDivEnd = divEnd[i];
#pragma unroll
    for (int j=0; j<_TA_; j++){
      int aid = aStartBlock + tid * _TA_ + j;
      if (aid < nAddress){
        ll_t adr = threadAddress[j];
        if (adr >= cDivStart && adr < cDivEnd){
          divs[aid] = i;
        }
      }
    } // end for j
  } // end for i
}
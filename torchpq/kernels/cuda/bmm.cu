#include "hip/hip_runtime.h"
extern "C"
__global__ void bmm_tn(
  const float* __restrict__ A,
  const float* __restrict__ B,
  float* __restrict__ C,
  int M, int N, int K
){
  int tid = threadIdx.x;     // thread idx
  int bid = blockIdx.z;      // batch idx

  // Neighboring blocks are grouped into PN x PM block groups in order to increase
  // L1 cache hit rate
  // There are ceil(M/PM) x ceil(N/PN) block groups in total.
  // Blocks within block groups are indexed with blockIdx.x % PN and blockIdx.x / PN
  int px = blockIdx.x % _PN_;
  int py = blockIdx.x / _PN_;
  int bDimX = (N + (128*_PN_) - 1) / (128*_PN_); 
  int bDimY = (M + (128*_PM_) - 1) / (128*_PM_); 
  int bIdxX = (blockIdx.y % bDimX) * _PN_ + px;
  int bIdxY = (blockIdx.y / bDimX) * _PM_ + py;
  int gStartx = bIdxX * 128;   // starting index of block on N axis
  int gStarty = bIdxY * 128;   // starting index of block on M axis
  if (gStartx > N || gStarty > M){
    return;
  }
  // These are used to re-arrange threads into different shapes
  // for example: (256) -> (16, 16) -> (8, 32) -> (32, 8)
  int vx = tid % 16;
  int vy = tid / 16;
  int wx = tid % 32; // thread idx in warp
  int wy = tid / 32; // warp id
  int dx = tid % 8;
  int dy = tid / 8;

  __shared__ _VOLATILE_ float aSmem1[8][128+4];
  __shared__ _VOLATILE_ float bSmem1[8][128+4];
  __shared__ _VOLATILE_ float aSmem2[8][128+4];
  __shared__ _VOLATILE_ float bSmem2[8][128+4];
  float aBuffer1[4];
  float bBuffer1[4];
  float aBuffer2[4];
  float bBuffer2[4];

  float8 cCache[8];
  init_cCache(cCache);

  // Load initial 16 x 128 tile of A and B to buffer1 and buffer2
  load_ab_tn(
    A, B, 
    aBuffer1, aBuffer2, bBuffer1, bBuffer2,
    bid, gStartx, gStarty, 0,
    M, N, K
  );

  // Number of main loop iterations is ceil(k/16)
  int nIt = (K + 16 - 1) / 16;
  #pragma unroll
  for (int itr=0; itr<nIt; itr++){
    int gStartk = itr * 16;

    #pragma unroll
    buffer2smem_tn(
      aSmem1, aSmem2, bSmem1, bSmem2,
      aBuffer1, aBuffer2, bBuffer1, bBuffer2
    );
    if (likely(itr < nIt - 1)){
      load_ab_tn(
        A, B, 
        aBuffer1, aBuffer2, bBuffer1, bBuffer2,
        bid, gStartx, gStarty, gStartk + 16,
        M, N, K
      );
    }
    // synchroznie threads in order make sure tiles of A and B are fully
    // loaded to shared memory.
    __syncthreads();

    // Each thread computes 8 x 8 matrix multiplication
    // Accumulate intermediate results in cCache
    // aSmem1, bSmem1, aSmem2, bSmem2 are consumed
    thread_matmul_v3(aSmem1, bSmem1, cCache, vx, vy);
    thread_matmul_v3(aSmem2, bSmem2, cCache, vx, vy);

    // synchronize threads to signal that shared memory is consumed.
    __syncthreads();
  }
  
  // At the end of main loop, store cCache to C
  //write_c(cCache, C, gStartx, gStarty, vx, vy, bid, M, N);
  write_c_v3(cCache, C, gStartx, gStarty, vx, vy, bid, M, N);
}

extern "C"
__global__ void bmm_nt(
  const float* __restrict__ A,
  const float* __restrict__ B,
  float* __restrict__ C,
  int M, int N, int K
){
  int tid = threadIdx.x;     // thread idx
  int bid = blockIdx.z;      // batch idx

  // Neighboring blocks are grouped into PN x PM block groups in order to increase
  // L1 cache hit rate
  // There are ceil(M/PM) x ceil(N/PN) block groups in total.
  // Blocks within block groups are indexed with blockIdx.x % PN and blockIdx.x / PN
  int px = blockIdx.x % _PN_;
  int py = blockIdx.x / _PN_;
  int bDimX = (N + (128*_PN_) - 1) / (128*_PN_); 
  int bDimY = (M + (128*_PM_) - 1) / (128*_PM_); 
  int bIdxX = (blockIdx.y % bDimX) * _PN_ + px;
  int bIdxY = (blockIdx.y / bDimX) * _PM_ + py;
  int gStartx = bIdxX * 128;   // starting index of block on N axis
  int gStarty = bIdxY * 128;   // starting index of block on M axis
  if (gStartx > N || gStarty > M){
    return;
  } K
  // These are used to re-arrange threads into different shapes
  // for example: (256) -> (16, 16) -> (8, 32) -> (32, 8)
  int vx = tid % 16;
  int vy = tid / 16;
  int wx = tid % 32; // thread idx in warp
  int wy = tid / 32; // warp id
  int dx = tid % 8;
  int dy = tid / 8;

  __shared__ _VOLATILE_ float aSmem1[8][128+4];
  __shared__ _VOLATILE_ float bSmem1[8][128+4];
  __shared__ _VOLATILE_ float aSmem2[8][128+4];
  __shared__ _VOLATILE_ float bSmem2[8][128+4];
  float aBuffer1[4];
  float bBuffer1[4];
  float aBuffer2[4];
  float bBuffer2[4];

  float8 cCache[8];
  init_cCache(cCache);

  // Load initial 16 x 128 tile of A and B to buffer1 and buffer2
  load_ab_nt(
    A, B, 
    aBuffer1, aBuffer2, bBuffer1, bBuffer2,
    bid, gStartx, gStarty, 0,
    M, N, K
  );

  // Number of main loop iterations is ceil(k/16)
  int nIt = (K + 16 - 1) / 16;
  #pragma unroll
  for (int itr=0; itr<nIt; itr++){
    int gStartk = itr * 16;

    buffer2smem_nt(
      aSmem1, aSmem2, bSmem1, bSmem2,
      aBuffer1, aBuffer2, bBuffer1, bBuffer2
    );
    if (likely(itr < nIt - 1)){
      load_ab_nt(
        A, B, 
        aBuffer1, aBuffer2, bBuffer1, bBuffer2,
        bid, gStartx, gStarty, gStartk + 16,
        M, N, K
      );
    }
    // synchroznie threads in order make sure tiles of A and B are fully
    // loaded to shared memory.
    __syncthreads();

    // Each thread computes 8 x 8 matrix multiplication
    // Accumulate intermediate results in cCache
    // aSmem1, bSmem1, aSmem2, bSmem2 are consumed
    thread_matmul_v3(aSmem1, bSmem1, cCache, vx, vy);
    thread_matmul_v3(aSmem2, bSmem2, cCache, vx, vy);

    // synchronize threads to signal that shared memory is consumed.
    __syncthreads();
  }
  
  // At the end of main loop, store cCache to C
  //write_c(cCache, C, gStartx, gStarty, vx, vy, bid, M, N);
  write_c_v3(cCache, C, gStartx, gStarty, vx, vy, bid, M, N);
}

extern "C"
__global__ void bmm_nn(
  const float* __restrict__ A,
  const float* __restrict__ B,
  float* __restrict__ C,
  int M, int N, int K
){
  int tid = threadIdx.x;     // thread idx
  int bid = blockIdx.z;      // batch idx

  // Neighboring blocks are grouped into PN x PM block groups in order to increase
  // L1 cache hit rate
  // There are ceil(M/PM) x ceil(N/PN) block groups in total.
  // Blocks within block groups are indexed with blockIdx.x % PN and blockIdx.x / PN
  int px = blockIdx.x % _PN_;
  int py = blockIdx.x / _PN_;
  int bDimX = (N + (128*_PN_) - 1) / (128*_PN_); 
  int bDimY = (M + (128*_PM_) - 1) / (128*_PM_); 
  int bIdxX = (blockIdx.y % bDimX) * _PN_ + px;
  int bIdxY = (blockIdx.y / bDimX) * _PM_ + py;
  int gStartx = bIdxX * 128;   // starting index of block on N axis
  int gStarty = bIdxY * 128;   // starting index of block on M axis
  if (gStartx > N || gStarty > M){
    return;
  }
  // These are used to re-arrange threads into different shapes
  // for example: (256) -> (16, 16) -> (8, 32) -> (32, 8)
  int vx = tid % 16;
  int vy = tid / 16;
  int wx = tid % 32; // thread idx in warp
  int wy = tid / 32; // warp id
  int dx = tid % 8;
  int dy = tid / 8;

  __shared__ _VOLATILE_ float aSmem1[8][128+4];
  __shared__ _VOLATILE_ float bSmem1[8][128+4];
  __shared__ _VOLATILE_ float aSmem2[8][128+4];
  __shared__ _VOLATILE_ float bSmem2[8][128+4];
  float aBuffer1[4];
  float bBuffer1[4];
  float aBuffer2[4];
  float bBuffer2[4];

  float8 cCache[8];
  init_cCache(cCache);

  // Load initial 16 x 128 tile of A and B to buffer1 and buffer2
  load_ab_nn(
    A, B, 
    aBuffer1, aBuffer2, bBuffer1, bBuffer2,
    bid, gStartx, gStarty, 0,
    M, N, K
  );

  // Number of main loop iterations is ceil(k/16)
  int nIt = (K + 16 - 1) / 16;
  #pragma unroll
  for (int itr=0; itr<nIt; itr++){
    int gStartk = itr * 16;

    #pragma unroll
    buffer2smem_nn(
      aSmem1, aSmem2, bSmem1, bSmem2,
      aBuffer1, aBuffer2, bBuffer1, bBuffer2
    );
    if (likely(itr < nIt - 1)){
      load_ab_nn(
        A, B, 
        aBuffer1, aBuffer2, bBuffer1, bBuffer2,
        bid, gStartx, gStarty, gStartk + 16,
        M, N, K
      );
    }
    // synchroznie threads in order make sure tiles of A and B are fully
    // loaded to shared memory.
    __syncthreads();

    // Each thread computes 8 x 8 matrix multiplication
    // Accumulate intermediate results in cCache
    // aSmem1, bSmem1, aSmem2, bSmem2 are consumed
    thread_matmul_v3(aSmem1, bSmem1, cCache, vx, vy);
    thread_matmul_v3(aSmem2, bSmem2, cCache, vx, vy);

    // synchronize threads to signal that shared memory is consumed.
    __syncthreads();
  }
  
  // At the end of main loop, store cCache to C
  //write_c(cCache, C, gStartx, gStarty, vx, vy, bid, M, N);
  write_c_v3(cCache, C, gStartx, gStarty, vx, vy, bid, M, N);
}

extern "C"
__global__ void bmm_tt(
  const float* __restrict__ A,
  const float* __restrict__ B,
  float* __restrict__ C,
  int M, int N, int K
){
  int tid = threadIdx.x;     // thread idx
  int bid = blockIdx.z;      // batch idx

  // Neighboring blocks are grouped into PN x PM block groups in order to increase
  // L1 cache hit rate
  // There are ceil(M/PM) x ceil(N/PN) block groups in total.
  // Blocks within block groups are indexed with blockIdx.x % PN and blockIdx.x / PN
  int px = blockIdx.x % _PN_;
  int py = blockIdx.x / _PN_;
  int bDimX = (N + (128*_PN_) - 1) / (128*_PN_); 
  int bDimY = (M + (128*_PM_) - 1) / (128*_PM_); 
  int bIdxX = (blockIdx.y % bDimX) * _PN_ + px;
  int bIdxY = (blockIdx.y / bDimX) * _PM_ + py;
  int gStartx = bIdxX * 128;   // starting index of block on N axis
  int gStarty = bIdxY * 128;   // starting index of block on M axis
  if (gStartx > N || gStarty > M){
    return;
  }
  // These are used to re-arrange threads into different shapes
  // for example: (256) -> (16, 16) -> (8, 32) -> (32, 8)
  int vx = tid % 16;
  int vy = tid / 16;
  int wx = tid % 32; // thread idx in warp
  int wy = tid / 32; // warp id
  int dx = tid % 8;
  int dy = tid / 8;

  __shared__ _VOLATILE_ float aSmem1[8][128+4];
  __shared__ _VOLATILE_ float bSmem1[8][128+4];
  __shared__ _VOLATILE_ float aSmem2[8][128+4];
  __shared__ _VOLATILE_ float bSmem2[8][128+4];
  float aBuffer1[4];
  float bBuffer1[4];
  float aBuffer2[4];
  float bBuffer2[4];

  float8 cCache[8];
  init_cCache(cCache);

  // Load initial 16 x 128 tile of A and B to buffer1 and buffer2
  load_ab_tt(
    A, B, 
    aBuffer1, aBuffer2, bBuffer1, bBuffer2,
    bid, gStartx, gStarty, 0,
    M, N, K
  );

  // Number of main loop iterations is ceil(k/16)
  int nIt = (K + 16 - 1) / 16;
  #pragma unroll
  for (int itr=0; itr<nIt; itr++){
    int gStartk = itr * 16;

    #pragma unroll
    buffer2smem_tt(
      aSmem1, aSmem2, bSmem1, bSmem2,
      aBuffer1, aBuffer2, bBuffer1, bBuffer2
    );
    if (likely(itr < nIt - 1)){
      load_ab_tt(
        A, B, 
        aBuffer1, aBuffer2, bBuffer1, bBuffer2,
        bid, gStartx, gStarty, gStartk + 16,
        M, N, K
      );
    }
    // synchroznie threads in order make sure tiles of A and B are fully
    // loaded to shared memory.
    __syncthreads();

    // Each thread computes 8 x 8 matrix multiplication
    // Accumulate intermediate results in cCache
    // aSmem1, bSmem1, aSmem2, bSmem2 are consumed
    thread_matmul_v3(aSmem1, bSmem1, cCache, vx, vy);
    thread_matmul_v3(aSmem2, bSmem2, cCache, vx, vy);

    // synchronize threads to signal that shared memory is consumed.
    __syncthreads();
  }
  
  // At the end of main loop, store cCache to C
  //write_c(cCache, C, gStartx, gStarty, vx, vy, bid, M, N);
  write_c_v3(cCache, C, gStartx, gStarty, vx, vy, bid, M, N);
}
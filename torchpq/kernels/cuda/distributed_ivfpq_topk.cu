#include "hip/hip_runtime.h"
#define _VOLATILE_ 
#define likely(x)      __builtin_expect(!!(x), 1)
#define unlikely(x)    __builtin_expect(!!(x), 0)
#define load(x)        __ldcg(x)
#define store(x, value) __stcs(x, value)
#ifndef INFINITY
#define INFINITY __int_as_float(0x7f800000)
#endif

typedef unsigned char uint8_t;
typedef long long ll_t;

typedef struct __builtin_align__(8)
{
  float value;
  float index;
} pair;

typedef struct __device_builtin__ __builtin_align__(_NCS_)
{
  uint8_t _VARNAMES_;
} _uint8n_t;

typedef union {
  _uint8n_t u8n;
  uint8_t val[_NCS_];
} uint8n_t;

__device__ __forceinline__ float atomicMax(float *address, float val)
{
  int ret = __float_as_int(*address);
  while(val > __int_as_float(ret))
  {
    int old = ret;
    if((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
        break;
  }
  return __int_as_float(ret);
}

__device__ __forceinline__ unsigned int bfe(
  unsigned int source,
  unsigned int bitIndex
) {
  unsigned int bit;
  asm volatile("bfe.u32 %0, %1, %2, %3;" : "=r"(bit) : "r"((unsigned int) source), "r"(bitIndex), "r"(1));
  return bit;
}

__device__ __forceinline__ void warp_comparator(
  float &value,
  float &index,
  const int stride,
  const int direction
){
  const float otherValue = __shfl_xor_sync(0xFFFFFFFF, value, stride);
  const float otherIndex = __shfl_xor_sync(0xFFFFFFFF, index, stride);
  bool condition = value < otherValue == direction;
  index = condition ? otherIndex : index;
  value = condition ? otherValue : value;
}

__device__ __forceinline__ void block_comparator(
  float &value,
  float &index,
  const int stride,
  const int direction,
  const int laneID,
  _VOLATILE_ float sMem[]
){
  float tempPrecomputed1 = sMem[laneID];
  float tempPrecomputed2 = sMem[_TPB_ + laneID];
  __syncthreads();

  sMem[laneID] = value;
  sMem[_TPB_ + laneID] = index;
  __syncthreads();

  float otherValue = sMem[laneID ^ stride];
  float otherIndex = sMem[_TPB_ + laneID ^ stride];
  __syncthreads();

  sMem[laneID] = tempPrecomputed1;
  sMem[_TPB_ + laneID] = tempPrecomputed2;
  __syncthreads();

  bool condition = value < otherValue == direction;
  value = condition ? otherValue : value;
  index = condition ? otherIndex : index;
  /*
  */
}

__device__ __forceinline__ void block_comparator_noop(
){
  __syncthreads();
  __syncthreads();
  __syncthreads();
  __syncthreads();
}

__device__ __forceinline__ void thread_comparator(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  const int direction
){
  bool condition = value > otherValue == direction;
  if (condition){
    value = otherValue;
    index = otherIndex;
  }
}

__device__ void bitonic_sort_2(
  float &value,
  float &index,
  int laneID
){
  warp_comparator(value, index, 1, bfe(laneID, 1) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_4(
  float &value,
  float &index,
  int laneID
){
  bitonic_sort_2(value, index, laneID);
  warp_comparator(value, index, 2, bfe(laneID, 2) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 2) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_8(
  float &value,
  float &index,
  int laneID
){
  bitonic_sort_4(value, index, laneID);
  warp_comparator(value, index, 4, bfe(laneID, 3) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 3) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 3) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_16(
  float &value,
  float &index,
  int laneID
){
  bitonic_sort_8(value, index, laneID);
  warp_comparator(value, index, 8, bfe(laneID, 4) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 4) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 4) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 4) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_32(
  float &value,
  float &index,
  int laneID
){
  bitonic_sort_16(value, index, laneID);
  warp_comparator(value, index, 16, bfe(laneID, 5) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 5) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 5) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 5) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 5) ^ bfe(laneID, 0));
}

__device__ void bitonic_sort_global_2(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

__device__ void bitonic_sort_global_4(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

__device__ void bitonic_sort_global_8(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

__device__ void bitonic_sort_global_16(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

__device__ void bitonic_sort_global_32(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  int laneID
) {
  if (_TPB_ - 32 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  }
}

#if _TPB_ >= 64
__device__ void bitonic_sort_64(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_32(value, index, laneID);
  block_comparator(value, index, 32, bfe(laneID, 6) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 6) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 6) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 6) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 6) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 6) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_64(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 64 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));

    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
  }
}

#if _TPB_ >= 128
__device__ void bitonic_sort_128(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_64(value, index, sMem, laneID);
  block_comparator(value, index, 64, bfe(laneID, 7) ^ bfe(laneID, 6), laneID, sMem);
  block_comparator(value, index, 32, bfe(laneID, 7) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 7) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 7) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 7) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 7) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 7) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_128(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 128 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 64, !bfe(laneID, 6), laneID, sMem);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
    block_comparator_noop();
  }
}

#if _TPB_ >= 256
__device__ void bitonic_sort_256(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_128(value, index, sMem, laneID);
  block_comparator(value, index, 128, bfe(laneID, 8) ^ bfe(laneID, 7), laneID, sMem);
  block_comparator(value, index, 64, bfe(laneID, 8) ^ bfe(laneID, 6), laneID, sMem);
  block_comparator(value, index, 32, bfe(laneID, 8) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 8) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 8) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 8) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 8) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 8) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_256(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 256 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 128, !bfe(laneID, 7), laneID, sMem);
    block_comparator(value, index, 64, !bfe(laneID, 6), laneID, sMem);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
  }
}

#if _TPB_ >= 512
__device__ void bitonic_sort_512(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_256(value, index, sMem, laneID);
  block_comparator(value, index, 256, bfe(laneID, 9) ^ bfe(laneID, 8), laneID, sMem);
  block_comparator(value, index, 128, bfe(laneID, 9) ^ bfe(laneID, 7), laneID, sMem);
  block_comparator(value, index, 64, bfe(laneID, 9) ^ bfe(laneID, 6), laneID, sMem);
  block_comparator(value, index, 32, bfe(laneID, 9) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 9) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 9) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 9) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 9) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 9) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_512(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 512 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 256, !bfe(laneID, 8), laneID, sMem);
    block_comparator(value, index, 128, !bfe(laneID, 7), laneID, sMem);
    block_comparator(value, index, 64, !bfe(laneID, 6), laneID, sMem);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
  }
}


#if _TPB_ >= 1024
__device__ void bitonic_sort_1024(
  float &value,
  float &index,
  _VOLATILE_ float sMem[],
  int laneID
){
  bitonic_sort_512(value, index, sMem, laneID);
  block_comparator(value, index, 512, bfe(laneID, 10) ^ bfe(laneID, 9), laneID, sMem);
  block_comparator(value, index, 256, bfe(laneID, 10) ^ bfe(laneID, 8), laneID, sMem);
  block_comparator(value, index, 128, bfe(laneID, 10) ^ bfe(laneID, 7), laneID, sMem);
  block_comparator(value, index, 64, bfe(laneID, 10) ^ bfe(laneID, 6), laneID, sMem);
  block_comparator(value, index, 32, bfe(laneID, 10) ^ bfe(laneID, 5), laneID, sMem);
  warp_comparator(value, index, 16, bfe(laneID, 10) ^ bfe(laneID, 4));
  warp_comparator(value, index, 8, bfe(laneID, 10) ^ bfe(laneID, 3));
  warp_comparator(value, index, 4, bfe(laneID, 10) ^ bfe(laneID, 2));
  warp_comparator(value, index, 2, bfe(laneID, 10) ^ bfe(laneID, 1));
  warp_comparator(value, index, 1, bfe(laneID, 10) ^ bfe(laneID, 0));
}
#endif
__device__ void bitonic_sort_global_1024(
  float &value,
  float &index,
  float otherValue,
  float otherIndex,
  _VOLATILE_ float sMem[],
  int laneID
) {
  if (_TPB_ - 1024 <= threadIdx.x){
    thread_comparator(value, index, otherValue, otherIndex, 0);
    block_comparator(value, index, 512, !bfe(laneID, 9), laneID, sMem);
    block_comparator(value, index, 256, !bfe(laneID, 8), laneID, sMem);
    block_comparator(value, index, 128, !bfe(laneID, 7), laneID, sMem);
    block_comparator(value, index, 64, !bfe(laneID, 6), laneID, sMem);
    block_comparator(value, index, 32, !bfe(laneID, 5), laneID, sMem);
    warp_comparator(value, index, 16, !bfe(laneID, 4));
    warp_comparator(value, index, 8, !bfe(laneID, 3));
    warp_comparator(value, index, 4, !bfe(laneID, 2));
    warp_comparator(value, index, 2, !bfe(laneID, 1));
    warp_comparator(value, index, 1, !bfe(laneID, 0));
  } else {
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
    block_comparator_noop();
  }
}

__device__ void load_precomputed_v1(
  const float *precomputed,
  _VOLATILE_ float *sMem,
  int nQuery
){
  const int tid = threadIdx.x;
  const int qid = blockIdx.x;
  if (tid < 256){
    #pragma unroll
    for (int i = 0; i < _M_; i++){
      #if _TPB_ >= 256
      int adr = (i * nQuery * _K_) + (qid * _K_) + (tid);
      sMem[i * _K_ + tid] = precomputed[adr];
      
      #else
      #pragma unroll
      for (int j = 0; j < _K_ / _TPB_; j++){
        int adr = (i * nQuery * _K_) + (qid * _K_) + (j * _TPB_ + tid);
        sMem[i * _K_ + j * _TPB_ + tid] = precomputed[adr];
      }
      #endif
    }
  }
  __syncthreads();
}

__device__ void load_precomputed_v2(
  const float *precomputed,
  _VOLATILE_ float *sMem,
  int iProbe, int nProbe
){
  const int tid = threadIdx.x;
  const int qid = blockIdx.x;
  if (tid < 256){
    #pragma unroll
    for (int i = 0; i < _M_; i++){
      #if _TPB_ >= 256
      // int adr = (i * nQuery * _K_) + (qid * _K_) + (tid);
      int adr = 
        (qid) * nProbe * _M_ * _K_ +\
        (iProbe) * _M_ * _K_ +\
        (i) * _K_ +\
        (tid);
      sMem[i * _K_ + tid] = precomputed[adr];
      
      #else
      #pragma unroll
      for (int j = 0; j < _K_ / _TPB_; j++){
        int adr = (qid) * nProbe * _M_ * _K_ +\
          (iProbe) * _M_ * _K_ +\
          (i) * _K_ +\
          (j * _TPB_ + tid);
        sMem[i * _K_ + j * _TPB_ + tid] = precomputed[adr];
      }
      #endif
    }
  }
  __syncthreads();
}

__device__ void load_precomputed_v3(
  const float* part1,
  const float* part2,
  _VOLATILE_ float *sMem,
  int iCell
){
  const int tid = threadIdx.x;
  const int qid = blockIdx.x;
  if (tid < 256){
    #pragma unroll
    for (int i = 0; i < _M_; i++){
      #if _TPB_ >= 256
      // int adr = (i * nQuery * _K_) + (qid * _K_) + (tid);
      int adr1 =\
        (qid) * _M_ * _K_ +\
        (i) * _K_ +\
        (tid);
      float precomputedValue = part1[adr1];

      int adr2 =\
        (iCell) * _M_ * _K_ +\
        (i) * _K_ +\
        (tid);
      sMem[i * _K_ + tid] = precomputedValue + part2[adr2];

      #else
      #pragma unroll
      for (int j = 0; j < _K_ / _TPB_; j++){
        int adr1 =\
          (qid) * _M_ * _K_ +\
          (i) * _K_ +\
          (j * _TPB_ + tid);
        float precomputedValue = part1[adr1];

        int adr2 =\
          (iCell) * _M_ * _K_ +\
          (i) * _K_ +\
          (j * _TPB_ + tid);
        sMem[i * _K_ + j * _TPB_ + tid] = precomputedValue + part2[adr2];
      }
      #endif
    }
  }
  __syncthreads();
}

__device__ void load_part1_to_cache(
  const float* part1,
  float part1Cache[_M_]
){
  const int tid = threadIdx.x;
  const int qid = blockIdx.x;
  if (tid < 256){
    #pragma unroll
    for (int i = 0; i < _M_; i++){
      #if _TPB_ >= 256
      int adr1 =\
        (qid) * _M_ * _K_ +\
        (i) * _K_ +\
        (tid);
      part1Cache[i] = part1[adr1];
      #endif
    }
  }
}

__device__ void load_part2_to_cache(
  const float* part2,
  float part2Cache[_M_],
  int iCell
){
  const int tid = threadIdx.x;
  const int qid = blockIdx.x;
  if (tid < 256){
    #pragma unroll
    for (int i = 0; i < _M_; i++){
      #if _TPB_ >= 256
      int adr2 =\
        (iCell) * _M_ * _K_ +\
        (i) * _K_ +\
        (tid);
      part2Cache[i] = part2[adr2];
      #endif
    }
  }
}

__device__ void store_precomputed_to_smem(
  float part1Cache[_M_],
  float part2Cache[_M_],
  _VOLATILE_ float *sMem
){
  const int tid = threadIdx.x;
  const int qid = blockIdx.x;
  __syncthreads();
  if (tid < 256){
    #pragma unroll
    for (int i = 0; i < _M_; i++){
      #if _TPB_ >= 256
      float part1Value = part1Cache[i];
      float part2Value = part2Cache[i];
      sMem[i * _K_ + tid] = part1Value + part2Value;
      #endif
    }
  }
  __syncthreads();
}

__device__ void load_consume_data(
  const uint8n_t* data,
  _VOLATILE_ float sMem[],
  float &value,
  int iN, int nData
){
  #pragma unroll
  for (int i = 0; i < _M_ / _NCS_; i++){
    uint8n_t threadData = data[(i * nData) + iN];
    float pre0 = sMem[(i * _NCS_ + 0) * _K_ + int(threadData.val[0]) ];
    float pre1 = sMem[(i * _NCS_ + 1) * _K_ + int(threadData.val[1]) ];
    float pre2 = sMem[(i * _NCS_ + 2) * _K_ + int(threadData.val[2]) ];
    float pre3 = sMem[(i * _NCS_ + 3) * _K_ + int(threadData.val[3]) ];
    value += pre0;
    value += pre1;
    value += pre2;
    value += pre3;
  }
}

__device__ void load_data(
  const uint8n_t* data,
  uint8n_t dataCache[_M_ / _NCS_],
  int iN, int nData
){
  #pragma unroll
  for (int i = 0; i < _M_ / _NCS_; i++){
    uint8n_t threadData = data[(i * nData) + iN];
    dataCache[i] = threadData;
  }
}

__device__ void consume_data(
  _VOLATILE_ float sMem[],
  uint8n_t dataCache[_M_ / _NCS_],
  float &value
){
  #pragma unroll
  for (int i = 0; i < _M_ / _NCS_; i++){
    uint8n_t threadData = dataCache[i];
    float pre0 = sMem[(i * _NCS_ + 0) * _K_ + int(threadData.val[0]) ];
    float pre1 = sMem[(i * _NCS_ + 1) * _K_ + int(threadData.val[1]) ];
    float pre2 = sMem[(i * _NCS_ + 2) * _K_ + int(threadData.val[2]) ];
    float pre3 = sMem[(i * _NCS_ + 3) * _K_ + int(threadData.val[3]) ];
    value += pre0;
    value += pre1;
    value += pre2;
    value += pre3;
  }
}

__device__ void sort(
  float &finalValue,
  float &finalIndex,
  float value,
  float index,
  _VOLATILE_ float sMem[],
  int nCandidates
){
  const int tid = threadIdx.x;
  #if _TPB_ == 32
  bitonic_sort_32(value, index, tid);

  #elif _TPB_ == 64
  bitonic_sort_64(value, index, sMem, tid);

  #elif _TPB_ == 128
  bitonic_sort_128(value, index, sMem, tid);

  #elif _TPB_ == 256
  bitonic_sort_256(value, index, sMem, tid);

  #elif _TPB_ == 512
  bitonic_sort_512(value, index, sMem, tid);

  #elif _TPB_ == 1024
  bitonic_sort_1024(value, index, sMem, tid);
  #endif
  
  switch (nCandidates){
    case 2:
      bitonic_sort_global_2(
        finalValue, finalIndex, value, index,
        tid);
        break;
    case 4:
      bitonic_sort_global_4(
        finalValue, finalIndex, value, index,
        tid);
        break;
    case 8:
      bitonic_sort_global_8(
        finalValue, finalIndex, value, index,
        tid);
        break;
    case 16:
      bitonic_sort_global_16(
        finalValue, finalIndex, value, index,
        tid);
        break;
    case 32:
      bitonic_sort_global_32(
        finalValue, finalIndex, value, index,
        tid);
        break;
    case 64:
      bitonic_sort_global_64(
        finalValue, finalIndex, value, index,
        sMem, tid);
        break;
    case 128:
      bitonic_sort_global_128(
        finalValue, finalIndex, value, index,
        sMem, tid);
        break;
    case 256:
      bitonic_sort_global_256(
        finalValue, finalIndex, value, index,
        sMem, tid);
        break;
    case 512:
      bitonic_sort_global_512(
        finalValue, finalIndex, value, index,
        sMem, tid);
        break;
    case 1024:
      bitonic_sort_global_1024(
        finalValue, finalIndex, value, index,
        sMem, tid);
        break;
  }
}

__device__ bool is_stack_empty(
  int stackSize
){
  return stackSize <= 0;
}

__device__ bool is_stack_full(
  int stackSize
){
  return stackSize >= _STACKCAP_ - 1;
}

__device__ void push_stack(
  pair stack[_STACKCAP_],
  pair newPair,
  int &stackSize
) {
  if (is_stack_full(stackSize)){
    return;
  } else {
    #pragma unroll
    for (int i = _STACKCAP_ - 1; i >= 1; i--){
      stack[i] = stack[i - 1];
    }
    stack[0] = newPair;
    stackSize ++;
  }
}

__device__ void pop_stack(
  pair stack[_STACKCAP_],
  pair &outPair,
  int &stackSize
) {
  if (is_stack_empty(stackSize)){
    return;
  } else {
    outPair = stack[0];
    #pragma unroll
    for (int i=0; i<_STACKCAP_-1; i++){
      stack[i] = stack[i+1];
    }
    stackSize--;
  }
}

__device__ void init_stack(
  pair stack[_STACKCAP_]
){
  pair emptyPair;
  emptyPair.value = -INFINITY;
  emptyPair.index = -1;
  #pragma unroll
  for (int i=0; i < _STACKCAP_; i++){
    stack[i] = emptyPair;
  }
}


extern "C"
__global__ void ivfpq_topk(
  const uint8n_t* __restrict__ data,
  const float* __restrict__ precomputed,
  const uint8_t* __restrict__ isEmpty,
  const ll_t* __restrict__ cellStart,
  const ll_t* __restrict__ cellSize,
  const ll_t* __restrict__ totalSize,
  const ll_t* __restrict__ nProbeList,
  float* __restrict__ gValue,
  ll_t* __restrict__ gIndex,
  int nData, int nQuery, int maxNProbe, int nCandidates
) {
  const int tid = threadIdx.x; // thread ID
  const int qid = blockIdx.x; // query ID
  const int nProbe = nProbeList[qid];

  pair stack[_STACKCAP_];
  int stackSize = 0;
  init_stack(stack);

  extern __shared__ _VOLATILE_ float sMem[]; // M * K
  load_precomputed_v1(precomputed, sMem, nQuery);
  float finalValue = -INFINITY;
  float finalIndex = -1;
  float minValue = -INFINITY;
  const ll_t threadTotalSize = totalSize[qid];
  const int nIter = (threadTotalSize + _TPB_ - 1) / _TPB_;
  int cCell = 0;
  int cCellStart = cellStart[qid * maxNProbe + cCell];
  int cCellSize = cellSize[qid * maxNProbe + cCell];
  int cCellEnd = cCellStart + cCellSize;
  int iN = cCellStart + tid;

  for (int i = 0; i < nIter; i++){
    while (iN >= cCellEnd){
      cCell ++;  // increment cell index by 1
      if (cCell >= nProbe)
        break;
      int pCellEnd = cCellEnd;
      int pCellStart = cCellStart;
      cCellStart = cellStart[qid * maxNProbe + cCell];
      if (cCellStart == pCellStart){
        continue;
      }
      cCellSize = cellSize[qid * maxNProbe + cCell];
      cCellEnd = cCellStart + cCellSize;
      iN = iN - pCellEnd + cCellStart;
    }
    pair newPair;
    newPair.value = -INFINITY;
    newPair.index = -1;
    int cIsEmpty = 0;
    if (likely(iN < cCellEnd)){
      newPair.value = 0.f;
      newPair.index = iN;
      cIsEmpty = isEmpty[iN];
      uint8n_t dataCache[_M_ / _NCS_];
      load_data(data, dataCache, iN, nData);
      consume_data(sMem, dataCache, newPair.value);
    }
    newPair.value = cIsEmpty == 0 ? newPair.value : -INFINITY;
    newPair.index = cIsEmpty == 0 ? newPair.index : -1;

    __syncthreads();
    float temp1, temp2;
    if (tid == 0){
      temp1 = sMem[0];
      temp2 = sMem[1];
      sMem[0] = 0;
    }
    __syncthreads();

    pair oldPair;
    oldPair.value = -INFINITY;
    oldPair.index = -1;
    if (is_stack_full(stackSize)){
      pop_stack(stack, oldPair, stackSize);
      if (oldPair.value > minValue){
        sMem[0] = 1;
      }
    }

    if (newPair.value > minValue){
      push_stack(stack, newPair, stackSize);
    }
    __syncthreads();

    if (sMem[0] > 0){      
      __syncthreads();
      sort(
        finalValue, finalIndex,
        oldPair.value, oldPair.index,
        sMem, nCandidates
      );
      __syncthreads();
      if (tid == _TPB_ - 1){
        sMem[1] = finalValue;
      }
      __syncthreads();
      minValue = sMem[1];
    }
    __syncthreads();
    if (tid == 0){
      sMem[0] = temp1;
      sMem[1] = temp2;
    }
    __syncthreads();
    iN += _TPB_;
  }

  sMem[0] = 0;
  __syncthreads();
  #pragma unroll
  for (int i=0; i<_STACKCAP_; i++){
    pair oldPair;
    oldPair.value = -INFINITY;
    oldPair.index = -1;
    if (!is_stack_empty(stackSize)){
      pop_stack(stack, oldPair, stackSize);
      if (oldPair.value > minValue){
        sMem[0] = 1;
      }
    }
    __syncthreads();

    if (sMem[0] > 0){      
      __syncthreads();
      sort(
        finalValue, finalIndex,
        oldPair.value, oldPair.index,
        sMem, nCandidates
      );
      __syncthreads();
      sMem[0] = 0;
      if (tid == _TPB_ - 1){
        sMem[1] = finalValue;
      }
      __syncthreads();
      minValue = sMem[1];
    }
    __syncthreads();
  }

  if (_TPB_ - nCandidates <= tid){
    const int writeAddress = (qid * nCandidates) + tid - ( _TPB_ - nCandidates);
    gValue[writeAddress] = finalValue;
    gIndex[writeAddress] = finalIndex;
  }
}

extern "C"
__global__ void ivfpq_topk_residual(
  const uint8n_t* __restrict__ data,
  const float* __restrict__ precomputed,
  const float* __restrict__ baseSims,
  const uint8_t* __restrict__ isEmpty,
  const ll_t* __restrict__ cellStart,
  const ll_t* __restrict__ cellSize,
  const ll_t* __restrict__ totalSize,
  const ll_t* __restrict__ nProbeList,
  float* __restrict__ gValue,
  ll_t* __restrict__ gIndex,
  int nData, int nQuery, int maxNProbe, int nCandidates
) {
  const int tid = threadIdx.x; // thread ID
  const int qid = blockIdx.x; // query ID
  const int nProbe = nProbeList[qid];

  extern __shared__ _VOLATILE_ float sMem[]; // M * K
  const ll_t threadTotalSize = totalSize[qid];
  float finalValue = -INFINITY;
  float finalIndex = -1;
  int cCellStart = -1;
  for (int cCell = 0; cCell < nProbe; cCell++){
    int pCellStart = cCellStart;
    cCellStart = cellStart[qid * maxNProbe + cCell];
    if (cCellStart == pCellStart){
      continue;
    }
    int cCellSize = cellSize[qid * maxNProbe + cCell];
    load_precomputed_v2(precomputed, sMem, cCell, maxNProbe);
    float cBaseSim = baseSims[qid * maxNProbe + cCell];
    int cCellEnd = cCellStart + cCellSize;
    int nIter = (cCellSize + _TPB_ - 1) / _TPB_;
    for (int iter = 0; iter < nIter; iter++ ){
      int iN = cCellStart + iter * _TPB_ + tid;
      float value;
      float index = iN;
      int cIsEmpty = 0;
      if (cCellStart <= iN && iN < cCellEnd){
        value = cBaseSim;
        cIsEmpty = isEmpty[iN];
        uint8n_t dataCache[_M_ / _NCS_];
        load_data(data, dataCache, iN, nData);
        consume_data(sMem, dataCache, value);
      } else {
        value = -INFINITY;
      }
      value = cIsEmpty == 0 ? value : -INFINITY;
      index = cIsEmpty == 0 ? index : -1;
      
      sort(
        finalValue, finalIndex,
        value, index,
        sMem, nCandidates
      );
    }
  }

  if (_TPB_ - nCandidates <= tid){
    const int writeAddress = (qid * nCandidates) + tid - ( _TPB_ - nCandidates);
    gValue[writeAddress] = finalValue;
    gIndex[writeAddress] = finalIndex;
  }
}

extern "C"
__global__ void ivfpq_topk_residual_precomputed(
  const uint8n_t* __restrict__ data,
  const float* __restrict__ part1,
  const float* __restrict__ part2,
  const ll_t* __restrict__ cells,
  const float* __restrict__ baseSims,
  const uint8_t* __restrict__ isEmpty,
  const ll_t* __restrict__ cellStart,
  const ll_t* __restrict__ cellSize,
  const ll_t* __restrict__ totalSize,
  const ll_t* __restrict__ nProbeList,
  float* __restrict__ gValue,
  ll_t* __restrict__ gIndex,
  int nData, int nQuery, int maxNProbe, int nCandidates
) {
  const int tid = threadIdx.x; // thread ID
  const int qid = blockIdx.x; // query ID
  const int nProbe = nProbeList[qid];

  pair stack[_STACKCAP_];
  int stackSize = 0;
  init_stack(stack);

  extern __shared__ _VOLATILE_ float sMem[]; // M * K
  const ll_t threadTotalSize = totalSize[qid];
  float finalValue = -INFINITY;
  float finalIndex = -1;
  float minValue = -INFINITY;
  float part1Cache[_M_];
  float part2Cache[_M_];
  load_part1_to_cache(part1, part1Cache);

  int nCellStart = cellStart[qid * maxNProbe];
  int nCellSize = cellSize[qid * maxNProbe];
  int nCellEnd = nCellStart + nCellSize;
  int iCell = cells[qid * maxNProbe];
  bool nCellRepeated = false;
  bool cCellRepeated = false;
  load_part2_to_cache(part2, part2Cache, iCell);

  for (int cCell = 0; cCell < nProbe; cCell++){
    int cCellStart = nCellStart;
    int cCellSize = nCellSize;
    int cCellEnd = nCellEnd;
    if (!cCellRepeated){
      store_precomputed_to_smem(part1Cache, part2Cache, sMem);
    }

    if (cCell < nProbe - 1){
      int tCellStart = cellStart[qid * maxNProbe + cCell + 1];
      if (tCellStart != cCellStart){
        nCellStart = tCellStart;
        nCellSize = cellSize[qid * maxNProbe + cCell + 1];
        nCellEnd = nCellStart + nCellSize;
        iCell = cells[qid * maxNProbe + cCell + 1];
        load_part2_to_cache(part2, part2Cache, iCell);
        nCellRepeated = false;
      } else {
        nCellRepeated = true;
      }
    }
    if (cCellRepeated){
      cCellRepeated = nCellRepeated;
      continue;
    }
    cCellRepeated = nCellRepeated;
    float cBaseSim = baseSims[qid * maxNProbe + cCell];
    int nIter = (cCellSize + _TPB_ - 1) / _TPB_;
    for (int iter = 0; iter < nIter; iter++ ){
      int iN = cCellStart + iter * _TPB_ + tid;
      pair newPair;
      newPair.value = -INFINITY;
      newPair.index = -1;
      int cIsEmpty = 0;
      if (iN < cCellEnd){
        newPair.value = cBaseSim;
        newPair.index = iN;
        cIsEmpty = isEmpty[iN];
        uint8n_t dataCache[_M_ / _NCS_];
        load_data(data, dataCache, iN, nData);
        consume_data(sMem, dataCache, newPair.value);
      }

      newPair.value = cIsEmpty == 0 ? newPair.value : -INFINITY;
      newPair.index = cIsEmpty == 0 ? newPair.index : -1;
      
      __syncthreads();
      float temp1, temp2;
      if (tid == 0){
        temp1 = sMem[0];
        temp2 = sMem[1];
        sMem[0] = 0;
      }
      __syncthreads();

      pair oldPair;
      oldPair.value = -INFINITY;
      oldPair.index = -1;
      if (is_stack_full(stackSize)){
        pop_stack(stack, oldPair, stackSize);
        if (oldPair.value > minValue){
          sMem[0] = 1;
        }
      }
      if (newPair.value > minValue){
        push_stack(stack, newPair, stackSize);
      }
      __syncthreads();
      if (sMem[0] > 0){
        __syncthreads();
        sort(
          finalValue, finalIndex,
          oldPair.value, oldPair.index,
          sMem, nCandidates
        );
        __syncthreads();
        if (tid == _TPB_ - 1){
          sMem[1] = finalValue;
        }
        __syncthreads();
        minValue = sMem[1];
      }
      __syncthreads();
      if (tid == 0){
        sMem[0] = temp1;
        sMem[1] = temp2;
      }
      __syncthreads();
    }
  }

  sMem[0] = 0;
  __syncthreads();
  for (int i=0; i < _STACKCAP_; i++){
    pair oldPair;
    oldPair.value = -INFINITY;
    oldPair.index = -1;
    if (!is_stack_empty(stackSize)){
      pop_stack(stack, oldPair, stackSize);
      if (oldPair.value > minValue){
        sMem[0] = 1;
      }
    }
    __syncthreads();

    if (sMem[0] > 0){
      __syncthreads();
      sort(
        finalValue, finalIndex,
        oldPair.value, oldPair.index,
        sMem, nCandidates
      );
      __syncthreads();
      sMem[0] = 0;
      if (tid == _TPB_ - 1){
        sMem[1] = finalValue;
      }
      __syncthreads();
      minValue = sMem[1];
    }
    __syncthreads();
  }

  if (_TPB_ - nCandidates <= tid){
    const int writeAddress = (qid * nCandidates) + tid - ( _TPB_ - nCandidates);
    gValue[writeAddress] = finalValue;
    gIndex[writeAddress] = finalIndex;
  }
}